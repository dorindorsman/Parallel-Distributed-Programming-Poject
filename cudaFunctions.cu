#include "hip/hip_runtime.h"
#include "cudaHeader.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_cooperative_groups.h>


//CHECK FUNCTION
void checkStatus(hipError_t cudaStatus, std::string err)
{
    if(cudaStatus != hipSuccess)
    {
        std::cout << err << std::endl;
        exit(1);

    }
}


__global__ void cudaCalculateBestScore (char* cuda_seq1,char* proc_cuda_seq,int lenProcSeq,int** cuda_pointer_relation,int cuda_offset,int* maxScoreCuda)
{
    //blockIdx.x - index of block
    //blockDim.x - size of block
     __shared__ int maxScoreMutant; 
     __shared__ int maxScore;  
    maxScore=INT_MIN;

    int currentOffset=blockIdx.x;
    int currentMutant=threadIdx.x+1; //+1 because the tread id start from 0 
	int currnetScoreCuda=0;

    //CALCULATE SCORE
    for (int i = 0; i < lenProcSeq; i++)
    {
            if (i < currentMutant){
                currnetScoreCuda += cuda_pointer_relation[proc_cuda_seq[i]- 'A'][cuda_seq1[i + currentOffset]- 'A'];
            }
            else{
                currnetScoreCuda += cuda_pointer_relation[proc_cuda_seq[i]- 'A'][cuda_seq1[i + currentOffset + 1]- 'A'];
            }
    }
 

    __syncthreads();
    //CHECK MAX VALUE INSIDE EACH BLOCK - ATOMIC FOR PREVENT DATA RACE 
    atomicMax(&maxScore,currnetScoreCuda);
	__syncthreads(); //TO MAKE SURE THE MAX 
		

    //UPDATE MAX MUTANT INSIDE EACH BLOCK OF THE SAME MAX VALUE FROM LINE 49 - ATOMIC FOR PREVENT DATA RACE    
    if(currnetScoreCuda==maxScore){
        atomicExch(&maxScoreMutant,currentMutant); 
    }   
    __syncthreads();//TO MAKE SURE THE MAX MUTANT
	   
    //CHECK MAX VALUE BETWEEN BLOCKS AND COPY - ATOMIC FOR PREVENT DATA RACE
	if(currentMutant==1){
		atomicMax(maxScoreCuda,maxScore);
	}

    __syncthreads();
    //COPY RESULT
	if(currentMutant==1 && maxScore==maxScoreCuda[0] ){
        atomicExch(&maxScoreCuda[1],currentOffset);
        atomicExch(&maxScoreCuda[2],maxScoreMutant);
    }

    __syncthreads();

}  


void  cudaKernal(score_seq* alignments_score, char* seq1,char* proc_seq,int cuda_offset,int relation_group[][ALPHABET_SIZE]){

    //VARIABLES
    char* proc_cuda_seq;
    char* cuda_seq1;
    int* maxScoreCuda;
    hipError_t cudaStatus;
    int numOfBlocks = cuda_offset;
    int lenSeq1=strlen(seq1);
    int lenProcSeq=strlen(proc_seq);
    size_t sizeSeq1 = (lenSeq1) * sizeof(char);
    size_t sizeProcSeq = (lenProcSeq) * sizeof(char);
    size_t sizeMaxScoreCuda = 3 * sizeof(int);



    cudaStatus = hipMalloc((void**)&proc_cuda_seq,sizeProcSeq); //ALLOCATION cuda proc_seq
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(proc_cuda_seq,proc_seq,sizeProcSeq,hipMemcpyHostToDevice); //MEMCOPY proc_seq
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");
    
	cudaStatus = hipMalloc((void**)&cuda_seq1,sizeSeq1); //ALLOCATION cuda seq1
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(cuda_seq1,seq1,sizeSeq1,hipMemcpyHostToDevice); //MEMCOPY seq1
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

	cudaStatus = hipMalloc((void**)&maxScoreCuda,sizeMaxScoreCuda); //ALLOCATION
    checkStatus(cudaStatus,"Cuda Malloc Failed!");

    int temp=INT_MIN;
    cudaStatus = hipMemcpy(maxScoreCuda,&temp,sizeof(int),hipMemcpyHostToDevice); //MEMCOPY FOR INITIALIZTION
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //ALLOCATION AND MEMCOPY FOR RELATION GROUP:
    int** cuda_relation_group=(int**)malloc(sizeof(int*)*ALPHABET_SIZE);
    for(int i=0; i<ALPHABET_SIZE;i++){
        cudaStatus=hipMalloc(&cuda_relation_group[i],sizeof(int)*ALPHABET_SIZE);
        checkStatus(cudaStatus,"Cuda Malloc Failed!");
        cudaStatus = hipMemcpy(cuda_relation_group[i],relation_group[i],sizeof(int)*ALPHABET_SIZE,hipMemcpyHostToDevice); 
        checkStatus(cudaStatus,"Cuda Malloc Failed!");
    }
    
    int** cuda_pointer_relation;
    cudaStatus=hipMalloc(&cuda_pointer_relation,ALPHABET_SIZE*sizeof(int*));
	checkStatus(cudaStatus,"Cuda Malloc Failed!");
	cudaStatus = hipMemcpy(cuda_pointer_relation,cuda_relation_group,sizeof(int*)*ALPHABET_SIZE,hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"Cuda Malloc Failed!");

    //CALCULATE BEST SCORE CUDA:
    cudaCalculateBestScore<<<numOfBlocks,lenProcSeq>>>(cuda_seq1,proc_cuda_seq,lenProcSeq,cuda_pointer_relation,cuda_offset,maxScoreCuda);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, "Cuda Failed!");

    //MEMCOPY RESULT:
    cudaStatus = hipMemcpy(&alignments_score->mutant,&maxScoreCuda[2],sizeof(int),hipMemcpyDeviceToHost); 
    cudaStatus = hipMemcpy(&alignments_score->offset,&maxScoreCuda[1],sizeof(int),hipMemcpyDeviceToHost); 
    cudaStatus = hipMemcpy(&alignments_score->alignment_score,&maxScoreCuda[0],sizeof(int),hipMemcpyDeviceToHost); 
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //FREE ALLOCATIONS:
    cudaStatus = hipFree(proc_cuda_seq);
    checkStatus(cudaStatus,"Cuda Free Failed!");
    cudaStatus = hipFree(cuda_seq1);
    checkStatus(cudaStatus,"Cuda Free Failed!");
	cudaStatus = hipFree(maxScoreCuda);
    checkStatus(cudaStatus,"Cuda Free Failed!");


    for(int i=0; i<ALPHABET_SIZE;i++){
        cudaStatus=hipFree(cuda_relation_group[i]);
        checkStatus(cudaStatus,"Cuda Free Failed!");
    }

	cudaStatus = hipFree(cuda_pointer_relation);
    checkStatus(cudaStatus,"Cuda Free Failed!");
	free(cuda_relation_group);

}


